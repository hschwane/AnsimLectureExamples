#include "hip/hip_runtime.h"
#include "mpUtils/mpUtils.h"
#include "mpUtils/mpGraphics.h"
#include "mpUtils/mpCuda.h"
#include "SimplexNoise.h"

#include "../particleRenderer/ParticleRenderer.h"
#include <random>

constexpr int wgsize = 256; // cuda work group size

void generateInitialConditions(std::vector<float4>& pos, std::vector<float4>& vel, std::vector<float4>& acc, int particleCount, int simulationMode) {
    std::random_device rd;
    std::default_random_engine rng(rd());

    pos.resize(particleCount);
    vel.resize(particleCount);
    acc.resize(particleCount);

    if(simulationMode == 0) { // spawn particles in a sphere with initial velocity
        std::uniform_real_distribution<float> dist(0.0f,1.0f);
        SimplexNoise simplexGenerator(0.6);
        for(int i = 0; i<particleCount; i++) {
            mpu::randUniformSphere(dist(rng),dist(rng),dist(rng), 1.0f, pos[i].x, pos[i].y, pos[i].z);

            vel[i] = make_float4(0,0,0,0);
            vel[i] += make_float4( cross(make_float3(pos[i]), make_float3(0,1.1,0)), 0);

            vel[i].x += 0.15f*simplexGenerator.fractal(3,pos[i].x,pos[i].y,pos[i].z);
            vel[i].y += 0.15f*simplexGenerator.fractal(3,pos[i].x+500,pos[i].y+500,pos[i].z+500);
            vel[i].z += 0.15f*simplexGenerator.fractal(3,pos[i].x-500,pos[i].y-500,pos[i].z-500);
        }
    } else if(simulationMode == 1) { // spawn particles in a rotating disc
        std::uniform_real_distribution<float> dist(0.0f,1.0f);
        for(int i = 0; i<particleCount; i++) {

            mpu::randUniformSphere(dist(rng),dist(rng),dist(rng), 2.0f, pos[i].x, pos[i].y, pos[i].z);
            while( length(pos[i]) < 0.4 || pos[i].y > 0.05 || pos[i].y < -0.05) {
                mpu::randUniformSphere(dist(rng),dist(rng),dist(rng), 2.0f, pos[i].x, pos[i].y, pos[i].z);
            }

            float3 direction = normalize( cross( make_float3(pos[i]), make_float3(0,1,0)));
            float v = sqrt( 50 / length(pos[i]));

            vel[i] = v * make_float4(direction,0);
        }
    }
}

__device__ float3 calcGravity(float3 posi, float3 posj, float epsilon2) {
    float3 rvec = posj - posi;
    float r2 = rvec.x*rvec.x + rvec.y*rvec.y + rvec.z*rvec.z + epsilon2;
    float r2e = r2 + epsilon2;
    float distSixth = r2e * r2e * r2e;
    float invDistCube = rsqrt(distSixth);
    return invDistCube * rvec;
}

__global__ void calcAcceleration(float4* pos, float4* vel, float4* acc, int particleCount, float epsilon2, int simulationMode)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i > particleCount)
        return;

    float3 posi = make_float3(pos[i]);
    float3 acci{0,0,0};

    for(int j=0; j<particleCount; j++) {
        float3 posj = make_float3(pos[j]);
        acci += calcGravity(posi,posj,epsilon2);
    }

    acci *= 1.0f/particleCount;
    if(simulationMode == 1) { // mode 1 means we have an accretion disk around a central body
        acci += 50*calcGravity(posi, float3{0,0,0}, epsilon2);
    }
    acc[i] = float4{acci.x,acci.y,acci.z,0};
}

__global__ void integrateLeapfrog(float4* pos, float4* vel, float4* acc, int particleCount, float dt)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i > particleCount)
        return;

    pos[i] = pos[i] + vel[i] * dt;
    vel[i] = vel[i] + acc[i] * dt;
}

__global__ void initializeLeapfrog(float4* pos, float4* vel, float4* acc, int particleCount, float dt)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i > particleCount)
        return;

    vel[i] = vel[i] + acc[i] * dt*0.5f;
}

void showSettingsWindow(int& particleCount, bool& isSimRunning, float& dt, float& simulatedTime, int& simulationMode, float& epsilon2,
                        std::vector<float4>& pos, std::vector<float4>& vel, std::vector<float4>& acc,
                        mpu::gph::Buffer<float4>& posBuffer, mpu::gph::Buffer<float4>& velBuffer,
                        mpu::gph::Buffer<float4>& accBuffer, mpu::GlBufferMapper<float4>& posMapper,
                        mpu::GlBufferMapper<float4>& velMapper, mpu::GlBufferMapper<float4>& accMapper,
                        ParticleRenderer& renderer)
{
    static std::once_flag onceFlag;
    static unsigned int uiParticleCount;
    static int uiSimulationMode;
    std::call_once(onceFlag, [&](){
       uiParticleCount = particleCount;
        uiSimulationMode = simulationMode;
    });

    using namespace mpu::gph;
    if(ImGui::Begin("Simulation"))
    {
        ImGui::Text("Simulated Time: %f units", simulatedTime);
        ImGui::DragFloat("DeltaT", &dt, 0.00005,0.0001,1.0,"%.4f");
        ImGui::ToggleButton("RunSimulation", &isSimRunning);

        ImGui::PushID("InitialConditions");
        if(ImGui::CollapsingHeader("Rendering Mode"))
        {
            ImGui::Text("Particles: %i",uiParticleCount);
            ImGui::SameLine();
            if (ImGui::Button("-")) {
                uiParticleCount = uiParticleCount >> 1;
            }
            ImGui::SameLine();
            if(ImGui::Button("+")) {
                uiParticleCount = uiParticleCount << 1;
            }

            ImGui::Combo("Mode",&uiSimulationMode,"Sphere\0Disk\0\0");

            if(ImGui::Button("Reset Simulation"))
            {
                simulatedTime = 0;
                isSimRunning = false;
                particleCount = uiParticleCount;
                simulationMode = uiSimulationMode;

                generateInitialConditions( pos, vel, acc, particleCount, simulationMode);
                posBuffer = mpu::gph::Buffer<float4>(pos);
                velBuffer = mpu::gph::Buffer<float4>(vel);
                accBuffer = mpu::gph::Buffer<float4>(acc);

                posMapper = mpu::mapBufferToCuda(posBuffer);
                velMapper = mpu::mapBufferToCuda(velBuffer);
                accMapper = mpu::mapBufferToCuda(accBuffer);

                posMapper.map();
                velMapper.map();
                accMapper.map();

                calcAcceleration<<<ceil(posMapper.size()/float(wgsize)),wgsize>>>(posMapper.data(), velMapper.data(), accMapper.data(), posMapper.size(), epsilon2, simulationMode);
                initializeLeapfrog<<<ceil(posMapper.size()/float(wgsize)),wgsize>>>(posMapper.data(), velMapper.data(), accMapper.data(), posMapper.size(),dt);

                posMapper.unmap();
                velMapper.unmap();
                accMapper.unmap();

                renderer.setBuffers(posBuffer,velBuffer);
            }
        }

        ImGui::PopID();
    }
    ImGui::End();
}

int main()
{
    // Setup window
    // --------------------------
    mpu::Log myLog( mpu::LogLvl::ALL, mpu::ConsoleSink());
    myLog.printHeader("graphicsTest", MPU_VERSION_STRING, MPU_VERSION_COMMIT, "");

    int width = 800;
    int height = 600;
    mpu::gph::Window window(width, height,"AnSim Lecture Example");
    ImGui::create(window);
    mpu::gph::enableVsync(true);

    ParticleRenderer renderer;

    window.addFBSizeCallback([&](int width, int height)
    {
        glViewport(0,0,width,height);
        renderer.updateProjection(width,height);
    });
    renderer.updateProjection(width,height);
    // --------------------------

    // Setup simulation
    // --------------------------
    int particleCount = 512; // number of particles in the simulation
    float dt = 0.005; // timestep
    float simulatedTime = 0.0f; // simulated time since beginning of simulation
    bool isSimRunning = false; // is simulation running right now
    float epsilon2 = 0.0005; // smoothing factor epsilon^2
    int simulationMode = 0; // the type of initial condition used

    std::vector<float4> pos;
    std::vector<float4> vel;
    std::vector<float4> acc;
    generateInitialConditions( pos, vel, acc, particleCount, simulationMode);

    mpu::gph::Buffer<float4> posBuffer(pos);
    mpu::gph::Buffer<float4> velBuffer(vel);
    mpu::gph::Buffer<float4> accBuffer(acc);
    renderer.setBuffers(posBuffer,velBuffer);

    mpu::GlBufferMapper posMapper = mpu::mapBufferToCuda(posBuffer);
    mpu::GlBufferMapper velMapper = mpu::mapBufferToCuda(velBuffer);
    mpu::GlBufferMapper accMapper = mpu::mapBufferToCuda(accBuffer);

    posMapper.map();
    velMapper.map();
    accMapper.map();

    calcAcceleration<<<ceil(posMapper.size()/float(wgsize)),wgsize>>>(posMapper.data(), velMapper.data(), accMapper.data(), posMapper.size(), epsilon2, simulationMode);
    initializeLeapfrog<<<ceil(posMapper.size()/float(wgsize)),wgsize>>>(posMapper.data(), velMapper.data(), accMapper.data(), posMapper.size(),dt);

    posMapper.unmap();
    velMapper.unmap();
    accMapper.unmap();
    // --------------------------

    // setup keybinding to start / stop simulation with "F"
    // --------------------------
    mpu::gph::Input::addButton("ToggleSim","", [&](mpu::gph::Window& w){
        isSimRunning = !isSimRunning;
    });
    mpu::gph::Input::mapKeyToInput("ToggleSim", GLFW_KEY_F);
    // --------------------------

    while (window.frameEnd(), mpu::gph::Input::update(), window.frameBegin())
    {
        // run simulation here
        if(isSimRunning) {
            posMapper.map();
            velMapper.map();
            accMapper.map();

            calcAcceleration<<<ceil(posMapper.size()/float(wgsize)),wgsize>>>(posMapper.data(), velMapper.data(), accMapper.data(), posMapper.size(), epsilon2, simulationMode);
            integrateLeapfrog<<<ceil(posMapper.size()/float(wgsize)),wgsize>>>(posMapper.data(), velMapper.data(), accMapper.data(), posMapper.size(),dt);
            simulatedTime += dt;

            posMapper.unmap();
            velMapper.unmap();
            accMapper.unmap();
        }

        showSettingsWindow(particleCount,isSimRunning,dt,simulatedTime,simulationMode,epsilon2,
                           pos,vel,acc,posBuffer,velBuffer,accBuffer,posMapper,velMapper,accMapper,renderer);
        mpu::gph::showBasicPerformanceWindow();
        renderer.showGui();
        renderer.render();
    }

    return 0;
}
